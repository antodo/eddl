// This file is part of EDDLL an European Distributed Deep Learning Library.
// Developed within the DeepHealth project.
// Boosting AI in Europe.
//
// The MIT License (MIT)
//
// Copyright (c) 2019 Roberto Paredes Palacios, <rparedes@dsic.upv.es>

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>

///////////////////////////////////////////
__global__ void fill(float *aptr,float *bptr,int t,int aini,int at,int bini,int bt,int tot,int inc)
{
  int i=blockIdx.x;
  int j=threadIdx.x;
  int k=blockIdx.y;

  int ap=(i*at)+((aini+j)*t)+k;
  int bp=(i*bt)+((bini+j)*t)+k;

  if (bp<tot)
    if (inc) bptr[bp]+=aptr[ap];
    else bptr[bp]=aptr[ap];

}


///////////////////////////////////////////
__global__ void sum_mat_row(float* a, float* b, float* c, long int rows, long int cols)
{
 long int ops=rows*cols;
 long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < ops)
   c[thread_id_x]=a[thread_id_x]+b[thread_id_x%cols];

}
///////////////////////////////////////////
__global__ void sum_mat_col(float* a, float* b, float* c, long int rows, long int cols)
{
 long int ops=rows*cols;
 long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < ops)
   c[thread_id_x]=a[thread_id_x]+b[thread_id_x/cols];

}
///////////////////////////////////////////
__global__ void set(float* a, float v, long int rows, long int cols)
{
 long int ops=rows*cols;
 long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < ops)
   a[thread_id_x]=v;

}

///////////////////////////////////////////
__global__ void mult(float* a, float v, long int rows, long int cols)
{
 long int ops=rows*cols;
 long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < ops)
   a[thread_id_x]*=v;

}
///////////////////////////////////////////
__global__ void el_mult(float* a, float *b, float *c, long int incC, long int rows, long int cols)
{
  long int ops=rows*cols;
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x < ops)
    if (incC) c[thread_id_x]+=a[thread_id_x]*b[thread_id_x];
    else c[thread_id_x]=a[thread_id_x]*b[thread_id_x];
}

///////////////////////////////////////////
__global__ void el_div(float* a, float *b, float *c, long int incC, long int rows, long int cols)
{
  long int ops=rows*cols;
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x < ops)
    if (incC) c[thread_id_x]+=a[thread_id_x]/(b[thread_id_x]);
    else c[thread_id_x]=a[thread_id_x]/(b[thread_id_x]);
}

///////////////////////////////////////////
__global__ void sum(float* a, float v, long int rows, long int cols)
{
 long int ops=rows*cols;
 long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < ops)
   a[thread_id_x]+=v;

}
///////////////////////////////////////////
__global__ void sum(float scA,float* a,float scB,float *b, float *c,long int incC, long int size)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x < size) {
    if (incC) c[thread_id_x]+=scA*a[thread_id_x]+scB*b[thread_id_x];
    else c[thread_id_x]=scA*a[thread_id_x]+scB*b[thread_id_x];
  }
}

///////////////////////////////////////////
__global__ void reduce_array_sum(float* array, long int ops, long int cols,float* result)
{
  extern __shared__ float arr_acc[];
  __shared__ float accumulate_result[1];

  long int thread_id_x = threadIdx.x +blockIdx.x*blockDim.x;
  float sum=0;
  arr_acc[thread_id_x]=0.0;

  if(thread_id_x==0)
  	accumulate_result[thread_id_x]=0.0;

  __syncthreads();
  if (thread_id_x<ops)
  {
  	for (long int i=0; i<cols;i++)
    		sum+=array[thread_id_x*cols+i];

  __syncthreads();
    	arr_acc[thread_id_x]=sum;
  __syncthreads();

  }

  if (thread_id_x==0)
  {
  	for (long int i=0; i<ops;i++)
      accumulate_result[thread_id_x]+=arr_acc[thread_id_x+i];

    result[thread_id_x]=accumulate_result[thread_id_x];//copy back to global memory from shared

  }
}

///////////////////////////////////////////
__global__ void log(float* a, long int rows, long int cols)
{
 long int ops=rows*cols;
 long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < ops)
   a[thread_id_x]=log(a[thread_id_x]);

}

///////////////////////////////////////////
__global__ void exp(float* a, long int rows, long int cols)
{
 long int ops=rows*cols;
 long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < ops)
   a[thread_id_x]=exp(a[thread_id_x]);

}

///////////////////////////////////////////
__global__ void sqrt(float* a, long int rows, long int cols)
{
 long int ops=rows*cols;
 long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < ops)
   a[thread_id_x]=sqrt(a[thread_id_x]);

}

///////////////////////////////////////////
__global__ void sqr(float* a, long int rows, long int cols)
{
 long int ops=rows*cols;
 long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < ops)
   a[thread_id_x]*=a[thread_id_x];

}

///////////////////////////////////////////
__global__ void mask(float* a, float v, long int rows, long int cols)
{
 long int ops=rows*cols;
 long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < ops)
   a[thread_id_x]=a[thread_id_x]<v;

}

///////////////////////////////////////////

__global__ void reduce_sum2D(float *a,float *b,long int rows,long int cols,long int axis)
{
  long int ops=rows*cols;
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x < ops)
    if (axis==0)
        b[thread_id_x%cols]+=a[thread_id_x];
    else
        b[thread_id_x/cols]+=a[thread_id_x];
}
///////////////////////////////////////////
__global__ void cent(float* a, float* b, float* c, long int size)
{

 long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < size){
   c[thread_id_x]=0;
   if (a[thread_id_x]) c[thread_id_x]-=a[thread_id_x]*log(b[thread_id_x]);
   if (a[thread_id_x]!=1.0) c[thread_id_x]-=(1.0-a[thread_id_x])*log(1.0-b[thread_id_x]);
  }
}


__global__ void accuracy(float* T, float* N,float* acc,long int cols, long int total_ops, int* MC_err)
{

long int thread_id_x = threadIdx.x + blockIdx.x*blockDim.x;
long int result_t=T[thread_id_x*cols];
float result_n=N[thread_id_x*cols];

long int row_max_t=0;
long int row_max_n=0;

long int aux_t;
float aux_n;
if (thread_id_x < total_ops)
{
  for(long int i=1;i<cols;i++)
  {
   aux_t=T[thread_id_x*cols+i];
   aux_n=N[thread_id_x*cols+i];

	if (aux_t>result_t)
	 {
  		result_t=aux_t;
      row_max_t=i;
   }
  if (aux_n>result_n)
	 {
		result_n=aux_n;
    row_max_n=i;
   }
  }

  acc[thread_id_x]=row_max_t;
  atomicAdd(MC_err,(long int)(row_max_t==row_max_n));
}

}

///////////////////////////////////////////
__global__ void relu(float *a,float *b,long int size)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x < size){
    if (a[thread_id_x]>0.0) b[thread_id_x]=a[thread_id_x];
    else b[thread_id_x]=0.0;
   }
}


__global__ void d_relu(float *d,float *i,float *pd,long int size)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x < size){
    if (i[thread_id_x]>0.0) pd[thread_id_x]=d[thread_id_x];
    else pd[thread_id_x]=0.0;
   }

}

///////////////////////////////////////////
__global__ void softmax(float* E,float* N,float* auxE ,long int sample_ndim, long int n_vals)
{
    float C_value=0;
    long int thread_id_x = threadIdx.x + blockIdx.x*blockDim.x;
    float maxCoef = E[thread_id_x*sample_ndim];
    float actualCoef = 0;
    if (thread_id_x<n_vals)
    {

	    for (long int cA = 1; cA < sample_ndim; cA++)
    		if (E[thread_id_x*sample_ndim+cA] > maxCoef)
    			 maxCoef=E[thread_id_x*sample_ndim+cA];

	    for (long int cA = 0; cA < sample_ndim; cA++)
  		{
  			actualCoef=expf(E[thread_id_x*sample_ndim+cA]-maxCoef);
  			auxE[thread_id_x*sample_ndim+cA]=actualCoef;
        C_value+=actualCoef;
  		}

      for (long int cA=0; cA < sample_ndim; cA++)
	       N[thread_id_x*sample_ndim+cA]=auxE[thread_id_x*sample_ndim+cA]/C_value;
    }

}
























///////////////////////////////////////////


///////////////////////////////////////////


///////////////////////////////////////////